#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <chrono>
#include "utils.cuh"
#include "types.cuh"

__global__ void maxmin_kernel(float *A, float *B, float *C_max, float *C_min, int M, int K, int N)
{
    int batch = blockIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N)
    {
        float max_val = -FLT_MAX;

        int g_idx = batch * M * N * K + row * N * K + col * K;
        // Almacenar todos los mínimos para esta posición
        for (int k = 0; k < K; ++k)
        {
            float a = A[batch * M * K + row * K + k];
            float b = B[batch * K * N + k * N + col];
            float min_ab = a < b ? a : b;

            // Actualizar máximo de los mínimos (esto va al tensor max)
            if (min_ab > max_val)
                max_val = min_ab;
            int c_min_idx = g_idx + k;
            C_min[c_min_idx] = min_ab;
        }
        // El tensor de máximos contiene el máximo de los mínimos (resultado tradicional de maxmin)
        int idx = batch * M * N + row * N + col;
        C_max[idx] = max_val;
    }
}

// Versión mejorada de maxmin que usa TensorResult y retorna tanto max como min
void maxmin(const TensorResult &tensor1, const TensorResult &tensor2,
            TensorResult &max_result, TensorResult &min_result,
            bool keep_in_device)
{
    // Validaciones básicas
    if (tensor1.data == nullptr || tensor2.data == nullptr)
    {
        fprintf(stderr, "Error: Los punteros de tensores no pueden ser nulos\n");
        return;
    }

    // Validar compatibilidad de dimensiones
    if (tensor1.N != tensor2.M || tensor1.batch != tensor2.batch)
    {
        fprintf(stderr, "Error: Dimensiones incompatibles entre tensores: %d y %d\n", tensor1.N, tensor2.M);
        return;
    }

    // Extraer dimensiones del tensor
    int batch = tensor1.batch;
    int M = tensor1.M;
    int K = tensor1.N;
    int N = tensor2.N;

    // Calcular tamaños de memoria
    size_t size_A = batch * M * K * sizeof(float);
    size_t size_B = batch * K * N * sizeof(float);
    size_t size_C_min = batch * M * N * K * sizeof(float);
    size_t size_C_max = batch * M * N * sizeof(float);

    float *d_A = nullptr, *d_B = nullptr, *d_C_max = nullptr, *d_C_min = nullptr;
    float *h_C_max = nullptr, *h_C_min = nullptr;

    // Asignar memoria para los resultados en device
    if (hipMalloc(&d_C_max, size_C_max) != hipSuccess)
    {
        fprintf(stderr, "Error: No se pudo asignar memoria para el resultado max en el dispositivo\n");
        return;
    }

    if (hipMalloc(&d_C_min, size_C_min) != hipSuccess)
    {
        fprintf(stderr, "Error: No se pudo asignar memoria para el resultado min en el dispositivo\n");
        hipFree(d_C_max);
        return;
    }

    // Inicializar memoria de resultados a cero
    if (hipMemset(d_C_max, 0, size_C_max) != hipSuccess ||
        hipMemset(d_C_min, 0, size_C_min) != hipSuccess)
    {
        fprintf(stderr, "Error: No se pudo inicializar memoria de resultados\n");
        hipFree(d_C_max);
        hipFree(d_C_min);
        return;
    }

    // Copiar datos al dispositivo si es necesario
    if (!tensor1.is_device_ptr)
    {
        // Si tensor1 está en host, copiarlo a device
        if (hipMalloc(&d_A, size_A) != hipSuccess)
        {
            fprintf(stderr, "Error: No se pudo asignar memoria para tensor1 en el dispositivo\n");
            hipFree(d_C_max);
            hipFree(d_C_min);
            return;
        }

        if (hipMemcpy(d_A, tensor1.data, size_A, hipMemcpyHostToDevice) != hipSuccess)
        {
            fprintf(stderr, "Error: No se pudo copiar tensor1 al dispositivo\n");
            hipFree(d_A);
            hipFree(d_C_max);
            hipFree(d_C_min);
            return;
        }
    }
    else
    {
        // Si ya está en el dispositivo, usarlo directamente
        d_A = tensor1.data;
    }

    // Hacer lo mismo para tensor2
    if (!tensor2.is_device_ptr)
    {
        // Si tensor2 está en host, copiarlo a device
        if (hipMalloc(&d_B, size_B) != hipSuccess)
        {
            fprintf(stderr, "Error: No se pudo asignar memoria para tensor2 en el dispositivo\n");
            if (!tensor1.is_device_ptr)
                hipFree(d_A);
            hipFree(d_C_max);
            hipFree(d_C_min);
            return;
        }

        if (hipMemcpy(d_B, tensor2.data, size_B, hipMemcpyHostToDevice) != hipSuccess)
        {
            fprintf(stderr, "Error: No se pudo copiar tensor2 al dispositivo\n");
            if (!tensor1.is_device_ptr)
                hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C_max);
            hipFree(d_C_min);
            return;
        }
    }
    else
    {
        // Si ya está en el dispositivo, usarlo directamente
        d_B = tensor2.data;
    }

    // Configurar dimensiones de grid y bloques
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
                 (M + blockDim.y - 1) / blockDim.y,
                 batch);

    // Ejecutar kernel que calcula tanto max como min
    maxmin_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C_max, d_C_min, M, K, N);
    hipDeviceSynchronize();
    // Verificar errores de lanzamiento
    hipError_t kernelError = hipGetLastError();
    if (kernelError != hipSuccess)
    {
        fprintf(stderr, "Error en la ejecución del kernel: %s\n",
                hipGetErrorString(kernelError));
        if (!tensor1.is_device_ptr)
            hipFree(d_A);
        if (!tensor2.is_device_ptr)
            hipFree(d_B);
        hipFree(d_C_max);
        hipFree(d_C_min);
        return;
    }

    // Liberar memoria temporal
    if (!tensor1.is_device_ptr)
        hipFree(d_A);
    if (!tensor2.is_device_ptr)
        hipFree(d_B);

    // Si queremos mantener el resultado en el dispositivo
    if (keep_in_device)
    {
        max_result = TensorResult(d_C_max, true, batch, M, N);
        min_result = TensorResult(d_C_min, true, batch, M, N, K);
        return;
    }

    // Si queremos el resultado en el host
    h_C_max = (float *)malloc(size_C_max);
    h_C_min = (float *)malloc(size_C_min);

    if (h_C_max == nullptr || h_C_min == nullptr)
    {
        fprintf(stderr, "Error: No se pudo asignar memoria para los resultados en host\n");
        if (h_C_max)
            free(h_C_max);
        if (h_C_min)
            free(h_C_min);
        hipFree(d_C_max);
        hipFree(d_C_min);
        return;
    }

    // Copiar de device a host
    if (hipMemcpy(h_C_max, d_C_max, size_C_max, hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(h_C_min, d_C_min, size_C_min, hipMemcpyDeviceToHost) != hipSuccess)
    {
        fprintf(stderr, "Error: No se pudo copiar los resultados al host\n");
        free(h_C_max);
        free(h_C_min);
        hipFree(d_C_max);
        hipFree(d_C_min);
        return;
    }

    // Liberar memoria de device
    hipFree(d_C_max);
    hipFree(d_C_min);

    // Asignar los resultados a las referencias
    max_result.data = h_C_max;
    max_result.is_device_ptr = false;
    max_result.batch = batch;
    max_result.M = M;
    max_result.N = N;
    max_result.K = 1;

    min_result.data = h_C_min;
    min_result.is_device_ptr = false;
    min_result.batch = batch;
    min_result.M = M;
    min_result.N = N;
    min_result.K = K;
}
